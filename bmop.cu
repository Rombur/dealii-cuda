/* -*- c-basic-offset:2; tab-width:2; indent-tabs-mode:nil -*-
 *
 */


#include <deal.II/base/subscriptor.h>
#include <deal.II/base/quadrature_lib.h>
#include <deal.II/base/function.h>
#include <deal.II/base/logstream.h>
#include <deal.II/base/timer.h>

#include <deal.II/lac/vector.h>
#include <deal.II/lac/constraint_matrix.h>
#include <deal.II/lac/vector_memory.h>

#include <deal.II/fe/fe_q.h>
#include <deal.II/fe/fe_values.h>

#include <deal.II/grid/tria.h>
#include <deal.II/grid/tria_accessor.h>
#include <deal.II/grid/tria_iterator.h>
#include <deal.II/grid/tria_boundary_lib.h>
#include <deal.II/grid/grid_generator.h>

#include <deal.II/numerics/vector_tools.h>

#include <deal.II/dofs/dof_handler.h>
#include <deal.II/dofs/dof_tools.h>

#include <fstream>
#include <sstream>


#include "matrix_free_gpu/defs.h"
#include "matrix_free_gpu/gpu_vec.h"
#include "matrix_free_gpu/cuda_utils.cuh"
#include "matrix_free_gpu/gpu_array.cuh"

#include "laplace_operator_gpu.h"

using namespace dealii;

// #define USE_HANGING_NODES
#define N_ITERATIONS 100

#ifdef DEGREE_FE
const unsigned int degree_finite_element = DEGREE_FE;
#else
const unsigned int degree_finite_element = 4;
#endif

#ifdef DIMENSION
const unsigned int dimension = DIMENSION;
#else
const unsigned int dimension = 3;
#endif

typedef double number;

//-------------------------------------------------------------------------
// problem
//-------------------------------------------------------------------------

template <int dim, int fe_degree>
class LaplaceProblem
{
public:
  LaplaceProblem ();
  void run ();

private:
  void setup_system ();
  void solve ();

  Triangulation<dim>               triangulation;
  FE_Q<dim>                        fe;
  DoFHandler<dim>                  dof_handler;
  ConstraintMatrix                 constraints;

  typedef LaplaceOperatorGpu<dim,fe_degree,number> SystemMatrixType;

  SystemMatrixType                 system_matrix;

  GpuVector<number>                src;
  GpuVector<number>                dst;

  double                           setup_time;
  ConditionalOStream               time_details;
  unsigned int                     n_iterations;
};



template <int dim, int fe_degree>
LaplaceProblem<dim,fe_degree>::LaplaceProblem ()
  :
  fe (fe_degree),
  dof_handler (triangulation),
  time_details (std::cout, false),
  n_iterations(N_ITERATIONS)
{}




template <int dim, int fe_degree>
void LaplaceProblem<dim,fe_degree>::setup_system ()
{
  Timer time;
  time.start ();
  setup_time = 0;

  system_matrix.clear();

  dof_handler.distribute_dofs (fe);

  std::cout << "Number of degrees of freedom: "
            << dof_handler.n_dofs()
            << std::endl;

  std::cout << "Number of elements: "
            << dof_handler.get_triangulation().n_active_cells()
            << std::endl;

  constraints.clear();
  VectorTools::interpolate_boundary_values (dof_handler,
                                            0,
                                            ZeroFunction<dim>(),
                                            constraints);
  DoFTools::make_hanging_node_constraints(dof_handler,constraints);
  constraints.close();
  setup_time += time.wall_time();
  time_details << "Distribute DoFs & B.C.     (CPU/wall) "
               << time() << "s/" << time.wall_time() << "s" << std::endl;
  time.restart();

  system_matrix.reinit (dof_handler, constraints);

  std::cout.precision(4);
  std::cout << "System matrix memory consumption:     "
            << system_matrix.memory_consumption()*1e-6
            << " MB."
            << std::endl;


  dst.reinit (system_matrix.n());
  src.reinit (system_matrix.n());

  setup_time += time.wall_time();
  time_details << "Setup matrix-free system   (CPU/wall) "
               << time() << "s/" << time.wall_time() << "s" << std::endl;
}



template <int dim, int fe_degree>
void LaplaceProblem<dim,fe_degree>::solve ()
{
  Timer time;

  // IC
  dst = 0.1;

  for(int i = 0; i < n_iterations; ++i) {
    dst.swap(src);

    system_matrix.vmult(dst,src);
  }

  hipDeviceSynchronize();

  time.stop();


  std::cout << "Time solve ("
            << n_iterations
            << " iterations)  (CPU/wall) " << time() << "s/"
            << time.wall_time() << "s\n";

  std::cout << "Per iteration "
            << time.wall_time() / n_iterations << "s\n";
}


template <int dim, int fe_degree>
void LaplaceProblem<dim,fe_degree>::run ()
{
  GridGenerator::hyper_cube (triangulation, 0., 1.);

  triangulation.refine_global (1);
  {
    typename Triangulation<dim>::active_cell_iterator
      it = triangulation.begin_active(),
      end = triangulation.end();
    for(; it != end; ++it) {
      Point<dim> p = it->center();

#ifdef USE_HANGING_NODES
          // if(p[0] > 0.5) it->set_refine_flag();
          bool ref = true;
          for(int d = 0; d < dim; ++d)
            ref = (p[d] > 0.5) && ref;
          if(ref) it->set_refine_flag();
#else
           it->set_refine_flag();
#endif

    }
  }

  triangulation.execute_coarsening_and_refinement();
  triangulation.refine_global (2);

  // set up roughly similar grids for different fe_degree (and scale up 2D
  // problem somewhat)
  if(dim == 2) {
    triangulation.refine_global (2);
  }
  else if(dim == 3) {

  }

  if(degree_finite_element ==1) {
    triangulation.refine_global (3);
  }
  else if(degree_finite_element ==2) {
    triangulation.refine_global (2);
  }
  else if(degree_finite_element ==3) {
    triangulation.refine_global (1);
  }
  else if(degree_finite_element ==4) {
    triangulation.refine_global (1);

  }


  setup_system ();
  solve ();
  std::cout << std::endl;
}



int main ()
{
  try
  {
    deallog.depth_console(0);
    printf("d: %d, p: %d\n",dimension,degree_finite_element);
    LaplaceProblem<dimension,degree_finite_element> laplace_problem;
    laplace_problem.run ();
  }
  catch (std::exception &exc)
  {
    std::cerr << std::endl << std::endl
              << "----------------------------------------------------"
              << std::endl;
    std::cerr << "Exception on processing: " << std::endl
              << exc.what() << std::endl
              << "Aborting!" << std::endl
              << "----------------------------------------------------"
              << std::endl;
    return 1;
  }
  catch (...)
  {
    std::cerr << std::endl << std::endl
              << "----------------------------------------------------"
              << std::endl;
    std::cerr << "Unknown exception!" << std::endl
              << "Aborting!" << std::endl
              << "----------------------------------------------------"
              << std::endl;
    return 1;
  }

  GrowingVectorMemory<GpuVector<number> >::release_unused_memory();

  return 0;
}
